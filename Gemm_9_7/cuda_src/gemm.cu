#include "hip/hip_runtime.h"
#include "gemm.h"
#include "gemm.cuh"


#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 12

#define PRINT_ERRORS
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
//extern "C"
void MatMul( float * a, float * b, float * c, unsigned n) {
		//hipError_t err = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		//printf("set flag: %s\n",hipGetErrorString(err));
		
		float * ad;
		float * bd;
		float * cd;
		const int array_size = n*n*sizeof(float);

		hipError_t err = hipMalloc((void**)&ad, array_size);
		
		#ifdef  PRINT_ERRORS
		printf("CUDA malloc A: %s\n",hipGetErrorString(err));
		#endif //PRINT_ERRORS
		
		err = hipMemcpy(ad, a, array_size, hipMemcpyHostToDevice);
		#ifdef  PRINT_ERRORS
		printf("Copy A to device: %s\n",hipGetErrorString(err));
		#endif //PRINT_ERRORS

		err = hipMalloc((void**)&bd, array_size);
		#ifdef  PRINT_ERRORS
		printf("CUDA malloc B: %s\n",hipGetErrorString(err));
		#endif //PRINT_ERRORS

		err = hipMemcpy(bd, b, array_size, hipMemcpyHostToDevice);
		#ifdef  PRINT_ERRORS
		printf("Copy B to device: %s\n",hipGetErrorString(err));
		#endif //PRINT_ERRORS

		err = hipMalloc((void**)&cd, array_size);
		#ifdef  PRINT_ERRORS
		printf("CUDA malloc C: %s\n",hipGetErrorString(err));
		#endif //PRINT_ERRORS

		err = hipMemcpy(cd, c, array_size, hipMemcpyHostToDevice);
		#ifdef  PRINT_ERRORS
		printf("Copy C to device: %s\n",hipGetErrorString(err));
		#endif //PRINT_ERRORS
	
		// Invoke kernel
		dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
		dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x,
						(n + dimBlock.y - 1) / dimBlock.y);
		
		MatMulKernel<<<dimGrid,dimBlock >>>(ad, bd, cd,n);

		#ifdef  PRINT_ERRORS
		err = hipDeviceSynchronize();
		printf("Run kernel: %s\n", hipGetErrorString(err));
		#endif //PRINT_ERRORS
		
		// Read C from device memory
		err = hipMemcpy(c, cd, array_size, hipMemcpyDeviceToHost);
		#ifdef  PRINT_ERRORS
		printf("Copy C off of device: %s\n",hipGetErrorString(err));
		#endif //PRINT_ERRORS
		
		hipFree(ad);
		hipFree(bd);
		hipFree(cd);
}
// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(float * ad, float * bd, float * cd, unsigned n) {
		// Each thread computes one element of C
		// by accumulating results into Cvalue
		float Cvalue = 0.0;
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		if(row >= n || col >= n) return;
		for (int e = 0; e < n; ++e)
			Cvalue += (ad[row+e*n]) * (bd[e + col*n]);
		cd[row + col*n] += Cvalue;
}
