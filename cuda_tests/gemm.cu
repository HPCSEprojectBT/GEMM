#include "hip/hip_runtime.h"
#include "gemm.h"
#include "gemm.cuh"


#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
//extern "C"
void MatMul(float * a, float * b, float * c, unsigned n) {
		// Load A and B to device memory
		//Matrix d_A;
		//d_A.width = A.width;
		//d_A.height = A.height;
		//size_t size = A.width * A.height * sizeof(float);
/*
		Matrix d_A;
		d_A.width = n;
		d_A.height = n;
		size_t size = n*n * sizeof(float);

		hipError_t err = hipMalloc(&d_A.elements, size);
		printf("CUDA malloc A matrix: %s\n",hipGetErrorString(err));
*/
		float * ad;
		float * bd;
		float * cd;
		const int array_size = n*n*sizeof(float);

		hipError_t err = hipMalloc((void**)&ad, array_size);
		printf("CUDA malloc A: %s\n",hipGetErrorString(err));

		err = hipMemcpy(ad, a, array_size, hipMemcpyHostToDevice);
		printf("Copy A to device: %s\n",hipGetErrorString(err));

		err = hipMalloc((void**)&bd, array_size);
		printf("CUDA malloc B: %s\n",hipGetErrorString(err));

		err = hipMemcpy(bd, b, array_size, hipMemcpyHostToDevice);
		printf("Copy B to device: %s\n",hipGetErrorString(err));

		err = hipMalloc((void**)&cd, array_size);
		printf("CUDA malloc C: %s\n",hipGetErrorString(err));

		err = hipMemcpy(cd, c, array_size, hipMemcpyHostToDevice);
		printf("Copy C to device: %s\n",hipGetErrorString(err));
	
		//printf("cd[7] reads %f\n",cd[7]);
		//printf("ad[7] reads %f\n",ad[7]);
		//printf("bd[7] reads %f\n",bd[7]);
		//printf("c[7] reads %f\n",c[7]);
		//printf("a[7] reads %f\n",a[7]);
		//printf("b[7] reads %f\n",b[7]);

		/*
		Matrix d_B;
		d_B.width = B.width;
		d_B.height = B.height;
		size = B.width * B.height * sizeof(float);

		err = hipMalloc(&d_B.elements, size);
		printf("CUDA malloc B: %s\n",hipGetErrorString(err));

		err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
		printf("Copy B to device: %s\n",hipGetErrorString(err));
		
		// Allocate C in device memory
		Matrix d_C;
		d_C.width = C.width;
		d_C.height = C.height;
		size = C.width * C.height * sizeof(float);

		err = hipMalloc(&d_C.elements, size);
		printf("CUDA malloc C: %s\n",hipGetErrorString(err));
		*/
		// Invoke kernel
		dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
		dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x,
						(n + dimBlock.y - 1) / dimBlock.y);
		MatMulKernel<<<dimGrid, dimBlock>>>(ad, bd, cd,n);

		err = hipDeviceSynchronize();
		printf("Run kernel: %s\n", hipGetErrorString(err));
		
		// Read C from device memory
		err = hipMemcpy(c, cd, array_size, hipMemcpyDeviceToHost);
		printf("Copy C off of device: %s\n",hipGetErrorString(err));
		
		//printf("cd[7] reads %f\n",cd[7]);
		//printf("ad[7] reads %f\n",ad[7]);
		//printf("bd[7] reads %f\n",bd[7]);
		//printf("c[7] reads %f\n",c[7]);
		//printf("a[7] reads %f\n",a[7]);
		//printf("b[7] reads %f\n",b[7]);
		// Free device memory
		hipFree(ad);
		hipFree(bd);
		// hipFree(d_C.elements);
}
// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(float * ad, float * bd, float * cd, unsigned n) {
		// Each thread computes one element of C
		// by accumulating results into Cvalue
		float Cvalue = 0.0;
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		if(row > n || col > n) return;
		for (int e = 0; e < n; ++e)
			Cvalue += (ad[row+e*n]) * (bd[e + col*n]);
		cd[row + col*n] = Cvalue;
}
